#include "hip/hip_runtime.h"
extern "C"
{

#include "ConvectionGPU2d.h"
#include "hip/hip_runtime.h"

static float *d_Q;
static float *d_temp;
static float *d_rhs;
static float *d_bcFlag;

__constant__ physics d_phys;  // phys properties
__constant__ float d_dx,d_dy; // spacial interval
__constant__ int d_ndim1,d_ndim2;

//private functions
float getTimeInterval(structMesh *mesh, physics *phys, float CFL);
float** BCFlag_create(structMesh *mesh);
__global__ void AdvectionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float *d_bcFlag);
__global__ void DiffusionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float *d_bcFlag);
__global__ void TiemAdvectionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float dt);
__global__ void SlopeLimiter(int Ntotal, float *d_Q, float *d_temp, float *d_bcFlag);

void ConvectionGPUSolve2d(structMesh *mesh, physics *phys, 
	float **c, float finalTime){

	float time = 0.0f;
    float CFL  = 0.1f;
    int Ntotal = mesh->ndim1*mesh->ndim2;
    // calculate dt
    float dt = getTimeInterval(mesh, phys, CFL);

    // GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_phys),  phys,        sizeof(physics));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dx),    &mesh->dx,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dy),    &mesh->dy,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_ndim1), &mesh->ndim1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_ndim2), &mesh->ndim2, sizeof(int));

    // allocate and copy GPU global memory
    int sz = Ntotal*sizeof(float);
    hipMalloc ((void**) &d_Q, sz);
    hipMalloc ((void**) &d_temp, sz);
    hipMalloc ((void**) &d_rhs, sz);
    hipMalloc ((void**) &d_bcFlag, sz);

    hipMemcpy(d_Q, c[0], Ntotal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_temp, c[0], Ntotal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rhs, c[0], Ntotal*sizeof(float), hipMemcpyHostToDevice);

    //set bcFlag
    float **bcFlag = BCFlag_create(mesh);
    SaveMatrix("bc.txt", bcFlag, mesh->ndim1, mesh->ndim2);
    hipMemcpy(d_bcFlag, bcFlag[0], Ntotal*sizeof(float), hipMemcpyHostToDevice);
    Matrix_free(bcFlag);

    // set threads number
    int ThreadsPerBlock = 256;
    int BolcksPerGrid   = (Ntotal+ThreadsPerBlock-1)/ThreadsPerBlock;

    // time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
#if 0
    int counter = 0;
    char filename[NAMELEN];
#endif
    printf("The spatical interval: dx=%f, dy=%f\n", mesh->dx, mesh->dy);
    printf("The time interval: dt=%f\n", dt);

    while(time<finalTime){
        if(time+dt<finalTime){
            time += dt;
        }else{
            dt   = finalTime - time;
            time = finalTime;
        }
    // int i;
    // for(i=0; i<10; i++){

        AdvectionGPU2d<<<BolcksPerGrid, ThreadsPerBlock>>>(Ntotal, d_Q, d_rhs, d_bcFlag);
        DiffusionGPU2d<<<BolcksPerGrid, ThreadsPerBlock>>>(Ntotal, d_Q, d_rhs, d_bcFlag);
        TiemAdvectionGPU2d<<<BolcksPerGrid, ThreadsPerBlock>>>(Ntotal, d_Q, d_rhs, dt);
        SlopeLimiter<<<BolcksPerGrid, ThreadsPerBlock>>>(Ntotal, d_Q, d_temp, d_bcFlag);

        hipMemcpy(d_Q, d_temp, Ntotal*sizeof(float), hipMemcpyDeviceToDevice);

#if 0 //write the variable to file to debug

        if(!(counter%1)){
            hipMemcpy(c[0], d_Q, Ntotal*sizeof(float), hipMemcpyDeviceToHost);
            snprintf(filename, NAMELEN, "result-%d.txt", counter);
            SaveMatrix(filename, c, mesh->ndim1, mesh->ndim2);
        }
        counter++;

#endif

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\nTime Usages: %f\n", elapsedTime);

    hipMemcpy(c[0], d_Q, Ntotal*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_Q);
    hipFree(d_temp);
    hipFree(d_rhs);
    hipFree(d_bcFlag);

    return;
}

__global__ void SlopeLimiter(int Ntotal, float *d_Q, float *d_temp, float *d_bcFlag){
    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    if(offset < Ntotal){
        int bottom = offset - 1;
        int top    = offset + 1;
        int left   = offset - d_ndim2;
        int right  = offset + d_ndim2;

        float flag   = d_bcFlag[offset];

        if(flag<0.5f){
            float c = d_Q[offset];
            float t = d_Q[top];
            float l = d_Q[left];
            float r = d_Q[right];
            float b = d_Q[bottom];

            d_temp[offset] = (t+l+b+r+c)/5.0f;
        }
    }

    return;
}

/* Advection terms */
__global__ void AdvectionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float *d_bcFlag){
    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    if(offset < Ntotal){
    
        int bottom = offset - 1;
        int top    = offset + 1;
        int left   = offset - d_ndim2;
        int right  = offset + d_ndim2;

        float flag   = d_bcFlag[offset];

        if(flag<0.5f){
            float c = d_Q[offset];
            float t = d_Q[top];
            float l = d_Q[left];
            float r = d_Q[right];
            float b = d_Q[bottom];

            if(d_phys.u > 0) //upwind scheme
                d_rhs[offset] = -d_phys.u*(c-l)/d_dx;
            else
                d_rhs[offset] = -d_phys.u*(r-c)/d_dx;

            if(d_phys.v > 0)
                d_rhs[offset] += -d_phys.v*(c-b)/d_dy;
            else
                d_rhs[offset] += -d_phys.v*(t-c)/d_dy;

            // d_rhs[offset] = -d_phys.u*(c-l)/d_dx -d_phys.v*(c-b)/d_dy;
        }

    }
    return;
}

__global__ void DiffusionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float *d_bcFlag){

    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    if(offset < Ntotal){
    
        int bottom = offset - 1;
        int top    = offset + 1;
        int left   = offset - d_ndim2;
        int right  = offset + d_ndim2;

        float flag   = d_bcFlag[offset];

        if(flag<0.5f){

            float c = d_Q[offset];
            float t = d_Q[top];
            float l = d_Q[left];
            float r = d_Q[right];
            float b = d_Q[bottom];

            d_rhs[offset] += -d_phys.Dx*(r+l-2*c)/d_dx/d_dx     
                        -d_phys.Dy*(t+b-2*c)/d_dy/d_dy;

        }

    }
    return;
}

/* Euler Advance scheme for Time Discretization */
__global__ void TiemAdvectionGPU2d(int Ntotal, float *d_Q, float *d_rhs, float dt){

    int offset = threadIdx.x + blockIdx.x * blockDim.x;
    float rhs  = d_rhs[offset];

    if(offset < Ntotal){
        d_Q[offset] += rhs*dt;
    }

    return;
}

/* Create boundary condition flag matrix */
float** BCFlag_create(structMesh *mesh){
    float **bcFlag = Matrix_create(mesh->ndim1, mesh->ndim2);
    int dim1, dim2;

    for(dim1=0; dim1<mesh->ndim1; dim1++){
        dim2=0;
        bcFlag[dim1][dim2] = 1.0f;
        bcFlag[dim2][dim1] = 1.0f;

        dim2=mesh->ndim2-1;
        bcFlag[dim1][dim2] = 1.0f;
        bcFlag[dim2][dim1] = 1.0f;
    }
    return bcFlag;
}

/**
 * @brief
 * Calculate the Time Interval.
 *
 */
float getTimeInterval(structMesh *mesh, physics *phy, float CFL){
    float dt;
    float dx = mesh->dx;
    float dy = mesh->dy;
    dt = dx/phy->u;
    dt = minf(dt, dy/phy->v);
    dt = minf(dt, dx*dx/phy->Dx);
    dt = minf(dt, dy*dy/phy->Dy);
    return dt*CFL;
}

}